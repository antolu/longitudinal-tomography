#include "hip/hip_runtime.h"
//
// Created by anton on 10/22/20.
//

#include "kick_and_drift.cuh"
#include <iostream>

#define THREADS_PER_BLOCK 512

//#include "sin.h"

#define cudaErrorCheck(exit_code) { gpuAssert((exit_code), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

extern "C" void GPU::kick_and_drift(
        double *__restrict__ xp,             // inn/out
        double *__restrict__ yp,             // inn/out
        double *__restrict__ denergy,         // inn
        double *__restrict__ dphi,            // inn
        const double *__restrict__ rf1v,      // inn
        const double *__restrict__ rf2v,      // inn
        const double *__restrict__ phi0,      // inn
        const double *__restrict__ deltaE0,   // inn
        const double *__restrict__ drift_coef,// inn
        const double * phi12,
        const double hratio,
        const int dturns,
        const int rec_prof,
        const int nturns,
        const int nparts,
        const int nprofs,
        const bool ftn_out) {

    double *d_xp, *d_yp;
    double *d_denergy, *d_dphi;
    double *d_rf1v, *d_rf2v, *d_phi0, *d_deltaE0, *d_drift_coef, *d_phi12;

    int size_xyp = nparts * nprofs * sizeof(double);
    int size_nparts = nparts * sizeof(double);
    int size_nturns = nturns * sizeof(double);

    cudaErrorCheck( hipMalloc((void **) &d_xp, size_xyp) );
    cudaErrorCheck( hipMalloc((void **) &d_yp, size_xyp) );

    cudaErrorCheck( hipMalloc((void **) &d_denergy, size_nparts) );
    cudaErrorCheck( hipMalloc((void **) &d_dphi, size_nparts) );

    cudaErrorCheck( hipMalloc((void **) &d_rf1v, size_nturns) );
    cudaErrorCheck( hipMalloc((void **) &d_rf2v, size_nturns) );
    cudaErrorCheck( hipMalloc((void **) &d_phi0, size_nturns) );
    cudaErrorCheck( hipMalloc((void **) &d_phi12, size_nturns) );
    cudaErrorCheck( hipMalloc((void **) &d_deltaE0, size_nturns) );
    cudaErrorCheck( hipMalloc((void **) &d_drift_coef, size_nturns) );

//    hipMemcpy(d_xp, xp, size_xyp, hipMemcpyHostToDevice);
//    hipMemcpy(d_yp, yp, size_xyp, hipMemcpyHostToDevice);

    cudaErrorCheck( hipMemcpy(d_denergy, denergy, size_nparts, hipMemcpyHostToDevice) );
    cudaErrorCheck( hipMemcpy(d_dphi, dphi, size_nparts, hipMemcpyHostToDevice) );

    cudaErrorCheck( hipMemcpy(d_rf1v, rf1v, size_nturns, hipMemcpyHostToDevice) );
    cudaErrorCheck( hipMemcpy(d_rf2v, rf2v, size_nturns, hipMemcpyHostToDevice) );
    cudaErrorCheck( hipMemcpy(d_phi0, phi0, size_nturns, hipMemcpyHostToDevice) );
    cudaErrorCheck( hipMemcpy(d_phi12, phi12, size_nturns, hipMemcpyHostToDevice) );
    cudaErrorCheck( hipMemcpy(d_deltaE0, deltaE0, size_nturns, hipMemcpyHostToDevice) );
    cudaErrorCheck( hipMemcpy(d_drift_coef, drift_coef, size_nturns, hipMemcpyHostToDevice) );

    k_d<<<nparts * nprofs/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_xp, d_yp, d_denergy, d_dphi, d_rf1v, d_rf2v,
            d_phi0, d_deltaE0, d_drift_coef,
            d_phi12, hratio, dturns, rec_prof, nturns, nparts, nprofs, ftn_out);
    cudaErrorCheck( hipPeekAtLastError() );

//    cudaErrorCheck( hipDeviceSynchronize() );

    cudaErrorCheck( hipMemcpy(xp, d_xp, size_xyp, hipMemcpyDeviceToHost) );
    cudaErrorCheck( hipMemcpy(yp, d_yp, size_xyp, hipMemcpyDeviceToHost) );

    cudaErrorCheck( hipFree(d_xp) );
    cudaErrorCheck( hipFree(d_yp) );
    cudaErrorCheck( hipFree(d_denergy) );
    cudaErrorCheck( hipFree(d_dphi) );
    cudaErrorCheck( hipFree(d_rf1v) );
    cudaErrorCheck( hipFree(d_rf2v) );
    cudaErrorCheck( hipFree(d_phi0) );
    cudaErrorCheck( hipFree(d_deltaE0) );
    cudaErrorCheck( hipFree(d_drift_coef) );
}


__global__ void GPU::k_d(double *__restrict__ xp,             // inn/out
                                    double *__restrict__ yp,             // inn/out
                                    double *__restrict__ denergy,         // inn
                                    double *__restrict__ dphi,            // inn
                                    const double *__restrict__ rf1v,      // inn
                                    const double *__restrict__ rf2v,      // inn
                                    const double *__restrict__ phi0,      // inn
                                    const double *__restrict__ deltaE0,   // inn
                                    const double *__restrict__ drift_coef,// inn
                                    const double * phi12,
                                    const double hratio,
                                    const int dturns,
                                    const int rec_prof,
                                    const int nturns,
                                    const int nparts,
                                    const int nprofs,
                                    const bool ftn_out) {

    int profile = rec_prof;
    int turn = rec_prof * dturns;

    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index >= nparts) {
        return;
    }

    xp[profile * nparts + index] = dphi[index];
    yp[profile * nparts + index] = denergy[index];

    // Upwards
    while (turn < nturns) {
        drift_up(dphi, denergy, drift_coef[turn], nparts, index);

        turn++;

        kick_up(dphi, denergy, rf1v[turn], rf2v[turn], phi0[turn], phi12[turn],
                     hratio, nparts, deltaE0[turn], index);

        if (turn % dturns == 0) {
            profile++;

            xp[profile * nparts + index] = dphi[index];
            yp[profile * nparts + index] = denergy[index];
//            if (ftn_out)
//                std::cout << " Tracking from time slice  "
//                          << rec_prof + 1 << " to  " << profile + 1
//                          << ",   0.000% went outside the image width."
//                          << std::endl;
        } //if
    } //while

    profile = rec_prof;
    turn = rec_prof * dturns;

    if (profile > 0) {

        // Going back to initial coordinates
        for (int i = 0; i < nparts; i++) {
            dphi[index] = xp[rec_prof * nparts + index];
            denergy[index] = yp[rec_prof * nparts + index];
        }

        // Downwards
        while (turn > 0) {
            kick_down(dphi, denergy, rf1v[turn], rf2v[turn], phi0[turn],
                           phi12[turn], hratio, nparts, deltaE0[turn], index);
            turn--;

            drift_down(dphi, denergy, drift_coef[turn], nparts, index);

            if (turn % dturns == 0) {
                profile--;

                for (int i = 0; i < nparts; i++) {
                    xp[profile * nparts + index] = dphi[index];
                    yp[profile * nparts + index] = denergy[index];
                }
//                if (ftn_out)
//                    std::cout << " Tracking from time slice  "
//                              << rec_prof + 1 << " to  " << profile + 1
//                              << ",   0.000% went outside the image width."
//                              << std::endl;
            }

        }//while
    }
}

__device__ void GPU::kick_up(const double *__restrict__ dphi,
                             double *__restrict__ denergy,
                             const double rfv1,
                             const double rfv2,
                             const double phi0,
                             const double phi12,
                             const double hratio,
                             const int nparts,
                             const double acc_kick,
                             const int index) {

    denergy[index] += rfv1 * sin(dphi[index] + phi0)
                      + rfv2 * sin(hratio * (dphi[index] + phi0 - phi12)) - acc_kick;
}

__device__ void GPU::kick_down(const double *__restrict__ dphi,
                               double *__restrict__ denergy,
                               const double rfv1,
                               const double rfv2,
                               const double phi0,
                               const double phi12,
                               const double hratio,
                               const int nparts,
                               const double acc_kick,
                               const int index) {

    denergy[index] -= rfv1 * sin(dphi[index] + phi0)
                      + rfv2 * sin(hratio * (dphi[index] + phi0 - phi12)) - acc_kick;
}

__device__ void GPU::drift_up(double *__restrict__ dphi,
                              const double *__restrict__ denergy,
                              const double drift_coef,
                              const int nparts,
                              const int index) {

    dphi[index] -= drift_coef * denergy[index];
}

__device__ void GPU::drift_down(double *__restrict__ dphi,
                                const double *__restrict__ denergy,
                                const double drift_coef,
                                const int nparts,
                                const int index) {

    dphi[index] += drift_coef * denergy[index];
}

__device__ void GPU::calc_xp_and_yp(double *__restrict__ xp,           // inn/out
                                    double *__restrict__ yp,           // inn/out
                                    const double *__restrict__ denergy, // inn
                                    const double *__restrict__ dphi,    // inn
                                    const double phi0,
                                    const double hnum,
                                    const double omega_rev0,
                                    const double dtbin,
                                    const double xorigin,
                                    const double dEbin,
                                    const double yat0,
                                    const int profile,
                                    const int nparts);
